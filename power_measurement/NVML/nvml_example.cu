/*
 * Copyright 1993-2019 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* This example demonstrates how to use NVML library with
 * C++11 multithreading to create GPU monitoring with a
 * high sampling rate by storing nvidia-smi data to RAM
 * and then writing the output to a file once computation
 * is complete.
 */

/* Includes, system */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* Includes, cuda */
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

/* Includes, custom */
#include "nvmlClass.h"

void calculate( int const & m, int const & n, int const & k, nvmlClass & nvml ) {

	float alpha = 1.0f, beta = 0.0f;
	int lda = m, ldb = k, ldc = m;

	size_t sizeA = m * k;
	size_t sizeB = k * n;
	size_t sizeC = m * n;

	hipblasHandle_t handle;

	/* Initialize CUBLAS */
	checkCudaErrors( hipblasCreate( &handle ) );

	/* Initialize CUBLAS */
	printf( "hipblasSgemm %dx%dx%d test running..\n", m, n, k );

	/* Allocate host memory for the matrices */
	thrust::host_vector<float> h_A( sizeA, 0 );
	thrust::host_vector<float> h_B( sizeB, 0 );
	thrust::host_vector<float> h_C( sizeC, 0 );
	thrust::host_vector<float> h_C_ref( sizeC, 0 );

	/* Fill the matrices with test data */
	/* Assume square matrices */
	for ( int i = 0; i < m * m; i++ ) {
		h_A[i] = rand( ) / static_cast<float>( RAND_MAX );
		h_B[i] = rand( ) / static_cast<float>( RAND_MAX );
	}

	/* Create thread to gather GPU stats */
	std::thread threadStart( &nvmlClass::getStats, &nvml );   // threadStart starts running

	/* Allocate device memory for the matrices */
	thrust::device_vector<float> d_A( h_A );
	thrust::device_vector<float> d_B( h_B );
	thrust::device_vector<float> d_C( sizeC, 0 );

	/* Retrieve raw pointer for device data */
	float * d_A_ptr = thrust::raw_pointer_cast( &d_A[0] );
	float * d_B_ptr = thrust::raw_pointer_cast( &d_B[0] );
	float * d_C_ptr = thrust::raw_pointer_cast( &d_C[0] );

	/* Performs operation using cublas */
	hipblasSgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A_ptr, lda, d_B_ptr, ldb, &beta, d_C_ptr, ldc );
	checkCudaErrors( hipDeviceSynchronize( ) );

	/* Allocate host memory for reading back the result from device memory */
	h_C = d_C;

	/* Create thread to kill GPU stats */
	/* Join both threads to main */
	std::thread threadKill( &nvmlClass::killThread, &nvml );
	threadStart.join( );
	threadKill.join( );

	/* Shutdown */
	checkCudaErrors( hipblasDestroy( handle ) );
}

/* Main */
int main( int argc, char **argv ) {

	int dev = findCudaDevice( argc, ( const char ** ) argv );
	if ( dev == -1 ) throw std::runtime_error( "!!!! No CUDA device found\n" );

	std::string filename = { "../data/gpuStats.csv" };

	// Create NVML class to retrieve GPU stats
	nvmlClass nvml( dev, filename );

	for ( int i = 512; i <= 1024; i *= 2 )
		calculate( i, i, i, nvml );

	return ( EXIT_SUCCESS );
}

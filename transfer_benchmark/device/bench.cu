
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

 #define CUDA_SAFE(call)                                                 \
    do {                                                                \
    hipError_t err = call;                                             \
    if (hipSuccess != err) {                                           \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.",     \
            __FILE__, __LINE__, hipGetErrorString(err));               \
        exit(EXIT_FAILURE);                                             \
    }                                                                   \
    } while (0)

#define START_M(name) \
    std::chrono::time_point<std::chrono::steady_clock> start_name, end_name; \
    start_name = std::chrono::steady_clock::now();

#define END_M(name) \
    end_name = std::chrono::steady_clock::now(); \
    unsigned long long elapsed_seconds_name = std::chrono::duration_cast<std::chrono::nanoseconds> (end_name-start_name).count(); \
    //std::cout << elapsed_seconds_name << "\n";

#define runs_ 1
__global__
void bench(double *in, double  *out, int size){
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int global_threads = blockDim.x * gridDim.x;
    for (
            int tuple_id = global_idx;
            tuple_id < size;
            tuple_id += global_threads
        )
    {
        out[tuple_id] = in[tuple_id];
    }
}

__global__
void bench_single(double *in, double  *out){
    int tuple_id = blockIdx.x * blockDim.x + threadIdx.x;
	out[tuple_id] = in[tuple_id];
}

int main(){
	int k = 45000 / 2;
	long n = 4096 *k;
	// long long n = 1024ll * 1024 * 1024 / sizeof(double);

	double *in_h, *out_h;
	double *in_d, *out_d;

	hipHostAlloc((void**)&in_h,  n*sizeof(double), hipHostMallocDefault);;
	hipHostAlloc((void**)&out_h, n*sizeof(double), hipHostMallocDefault);;


	CUDA_SAFE(hipMalloc((void**)&in_d,  n*sizeof(double)));
	CUDA_SAFE(hipMalloc((void**)&out_d, n*sizeof(double)));

	// cudaMallocManaged((void**)&in_h,  n*sizeof(double));
	// cudaMallocManaged((void**)&out_h, n*sizeof(double));
	
	double gb = (double) n* sizeof(double) / 1024 / 1024 / 1024 * runs_;
	unsigned long long  total = 0;

	int runs = 4;
	for (int j = 0; j < runs; j++){
	/*	
		for (long i = 0; i < n; i++){
			in_h[i] = 1;
			out_h[i] = 0;
		}
*/
		for (int i = 0; i < n; i++) {
			in_h[i] = i;
			out_h[i] = i;
		}
	
		CUDA_SAFE(hipMemcpy(in_d, in_h, n* sizeof(double), hipMemcpyHostToDevice));
		CUDA_SAFE(hipMemcpy(out_d, out_h, n* sizeof(double), hipMemcpyHostToDevice));

		START_M()
		bench<<<128 * 32, 1024>>>(in_d, out_d, n);
		//bench_single<<<n/1024, 1024>>>(in_d, out_d);
		//bench<<<4*k, 1024>>>(in_h, out_h);
		CUDA_SAFE(hipDeviceSynchronize());
		END_M()
		
		CUDA_SAFE(hipMemcpy(in_h, in_d, n* sizeof(double), hipMemcpyDeviceToHost));
		CUDA_SAFE(hipMemcpy(out_h, out_d, n* sizeof(double), hipMemcpyDeviceToHost));

		total += elapsed_seconds_name;
		/*for (double i = 0; i < n; i++)
			if (out_h[i] != 1) prdoublef("Error\n");*/

	}
	//double sec  = (total / runs) / (1000ull *  1000ull * 1000ull);
	double nsec  = (total / runs);

	//std::cout << "sec " << sec << " gb " << gb << " gb/s " << gb/sec<<"\n";
	std::cout << "nsec " << nsec << " gb " << gb << " gb/s " << (2 * n*sizeof(double))/nsec<<"\n";
}

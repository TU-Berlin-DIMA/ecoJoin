
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

 #define CUDA_SAFE(call)                                                 \
    do {                                                                \
    hipError_t err = call;                                             \
    if (hipSuccess != err) {                                           \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.",     \
            __FILE__, __LINE__, hipGetErrorString(err));               \
        exit(EXIT_FAILURE);                                             \
    }                                                                   \
    } while (0)

#define START_M(name) \
    std::chrono::time_point<std::chrono::steady_clock> start_name, end_name; \
    start_name = std::chrono::steady_clock::now();

#define END_M(name) \
    end_name = std::chrono::steady_clock::now(); \
    unsigned long long elapsed_seconds_name = std::chrono::duration_cast<std::chrono::nanoseconds> (end_name-start_name).count(); \
    //std::cout << elapsed_seconds_name << "\n";

#define runs_ 1
__global__
void bench(int *in, int  *out, int size){
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int global_threads = blockDim.x * gridDim.x;
    for (
            int tuple_id = global_idx;
            tuple_id < size;
            tuple_id += global_threads
        )
    {
        //out[tuple_id] = in[tuple_id];
		atomicCAS(&out[tuple_id],tuple_id,in[tuple_id]);
    }
}

__global__
void bench_single(int *in, int  *out){
    int tuple_id = blockIdx.x * blockDim.x + threadIdx.x;
	out[tuple_id] = in[tuple_id];
}

int main(){
	int k = 45000 / 2;
	long n = 4096 *k;
	// long long n = 1024ll * 1024 * 1024 / sizeof(int);

	int *in_h, *out_h;
	int *in_d, *out_d;

	hipHostAlloc((void**)&in_h,  n*sizeof(int), hipHostMallocDefault);;
	hipHostAlloc((void**)&out_h, n*sizeof(int), hipHostMallocDefault);;


	CUDA_SAFE(hipMalloc((void**)&in_d,  n*sizeof(int)));
	CUDA_SAFE(hipMalloc((void**)&out_d, n*sizeof(int)));

	// cudaMallocManaged((void**)&in_h,  n*sizeof(int));
	// cudaMallocManaged((void**)&out_h, n*sizeof(int));
	
	double  gb = (double) n* sizeof(int) / 1024 / 1024 / 1024 * runs_;
	unsigned long long  total = 0;

	int runs = 20;
	for (int j = 0; j < runs; j++){
	/*	
		for (long i = 0; i < n; i++){
			in_h[i] = 1;
			out_h[i] = 0;
		}
*/
		for (int i = 0; i < n; i++) {
			in_h[i] = i;
			out_h[i] = i;
		}
	
		CUDA_SAFE(hipMemcpy(in_d, in_h, n* sizeof(int), hipMemcpyHostToDevice));
		CUDA_SAFE(hipMemcpy(out_d, out_h, n* sizeof(int), hipMemcpyHostToDevice));

		START_M()
		bench<<<128 * 32, 1024>>>(in_d, out_d, n);
		//bench_single<<<n/1024, 1024>>>(in_d, out_d);
		//bench<<<4*k, 1024>>>(in_h, out_h);
		CUDA_SAFE(hipDeviceSynchronize());
		END_M()
		
		CUDA_SAFE(hipMemcpy(in_h, in_d, n* sizeof(int), hipMemcpyDeviceToHost));
		CUDA_SAFE(hipMemcpy(out_h, out_d, n* sizeof(int), hipMemcpyDeviceToHost));

		total += elapsed_seconds_name;
		/*for (int i = 0; i < n; i++)
			if (out_h[i] != 1) printf("Error\n");*/

	}
	//int sec  = (total / runs) / (1000ull *  1000ull * 1000ull);
	double nsec  = (total / runs);

	//std::cout << "sec " << sec << " gb " << gb << " gb/s " << gb/sec<<"\n";
	std::cout << "nsec " << nsec << " gb " << gb << " gb/s " << (2 * n*sizeof(int))/nsec<<"\n";
}

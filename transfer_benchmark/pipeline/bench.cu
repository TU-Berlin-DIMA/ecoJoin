#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <omp.h>
#include "benchmark_helper.h"

 #define CUDA_SAFE(call)                                                 \
    do {                                                                \
    hipError_t err = call;                                             \
    if (hipSuccess != err) {                                           \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.",     \
            __FILE__, __LINE__, hipGetErrorString(err));               \
        exit(EXIT_FAILURE);                                             \
    }                                                                   \
    } while (0)

#define START_M(name) \
    std::chrono::time_point<std::chrono::system_clock> start_name, end_name; \
    start_name = std::chrono::system_clock::now();

#define END_M(name) \
    end_name = std::chrono::system_clock::now(); \
    long elapsed_seconds_name = std::chrono::duration_cast<std::chrono::nanoseconds> (end_name-start_name).           count(); \
    //std::cout << elapsed_seconds_name << "\n";

#define runs_ 1
__global__
void bench(int *in, int  *out, int size){
	unsigned global_idx = blockIdx.x * blockDim.x + threadIdx.x;
	const uint32_t global_threads = blockDim.x * gridDim.x;
    for (
            unsigned tuple_id = global_idx;
            tuple_id < size;
            tuple_id += global_threads
        )
    {
		out[tuple_id] = in[tuple_id];
    }
}


int main(){
	int k = 45000;
	long n = 4096 *k;

	int *in_h, *out_h;
	int *in_d, *out_d;

//	hipHostAlloc((void**)&in_h,  n*sizeof(int), hipHostMallocMapped);;
//	hipHostAlloc((void**)&out_h, n*sizeof(int), hipHostMallocMapped);;
	
	hipHostAlloc((void**)&in_h,  n*sizeof(int), hipHostMallocDefault);;
	hipHostAlloc((void**)&out_h, n*sizeof(int), hipHostMallocDefault);;
	

	CUDA_SAFE(hipMalloc((void**)&in_d,  n*sizeof(int)));
	CUDA_SAFE(hipMalloc((void**)&out_d, n*sizeof(int)));

	//hipMallocManaged((void**)&in_h,  n*sizeof(int));
	//hipMallocManaged((void**)&out_h, n*sizeof(int));
	

	int streamNumber = 5;
    hipStream_t *streams = (hipStream_t *)malloc(streamNumber * sizeof(hipStream_t));
    for (int i = 0; i < streamNumber; i++)
      CUDA_SAFE(hipStreamCreate(&streams[i]));

    int block = 1024 * 1024 *128; //  Blocksize in byte
    block = 1024 * 1024 * 32; //  Blocksize in byte
    long sizeByte =  n* sizeof(int);
    INIT_ASYNC(block, int)


	double gb = (double) n* sizeof(int) / 1024 / 1024 / 1024 * runs_;
	long total = 0;

	int runs = 1;
	for (int j = 0; j < runs; j++){
		
		for (long i = 0; i < n; i++){
			in_h[i] = 1;
			out_h[i] = 0;
		}

        START_M()
#pragma omp parallel for
		for (int k = 0; k < runs_; k++) {
			for (int i = 0; i < blockNumber; i++) {
				int offset = i * elementsPerBlock;
				CUDA_SAFE(hipMemcpyAsync(&in_d[offset], &in_h[offset], blockByte,
										hipMemcpyHostToDevice,
										streams[currentStream]));
				bench<<<128*32,64, 0, streams[currentStream]>>>(&in_d[offset], &out_d[offset], blockByte/sizeof(int));
				CUDA_SAFE(hipMemcpyAsync(&out_h[offset], &out_d[offset],
									blockByte, hipMemcpyDeviceToHost,
									streams[currentStream]));
				currentStream = ++currentStream % streamNumber;
			}
		}
        CUDA_SAFE(hipDeviceSynchronize());
        END_M()
		
		std::cout << "blockNumer: " << blockNumber  << " ElementPerBlock: " << blockByte/sizeof(int) << std::endl;
		total += elapsed_seconds_name;
		//for (int i = 0; i < n; i++)
		//	if (out_h[i] != 1) printf("Error\n");

	}
	double sec  = (double)(total / runs) / 1000000000;

	std::cout << "sec " << sec << " gb " << gb << " gb/s " << gb/sec<<"\n";

}

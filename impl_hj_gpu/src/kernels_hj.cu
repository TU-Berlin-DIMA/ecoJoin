#include "hip/hip_runtime.h"
#include <inttypes.h>
#include <assert.h>

#include "master.h"
#include "worker.h"
#include "MurmurHash.h"


__device__ inline 
int bindex(int b) { return b / (sizeof(unsigned)*8); }

__device__ inline 
int boffset(int b) { return b % (sizeof(unsigned)*8); }
//int boffset(int b) { return b >> 5; }

// FIXME: atomicOr ???
__device__ inline
void set_bit(int b, unsigned *array) { 
    array[bindex(b)] |= (1 << boffset(b));
}

__device__ inline
void clear_bit(int b, unsigned *array) { 
    array[bindex(b)] &= ~(1 << (boffset(b)));
}

__device__  inline
bool get_bit(int b, unsigned *array) { 
    return array[bindex(b)] & (1 << boffset(b));
}

// Struct definitions
struct chunk_R{
	long t_ns; /* timestamp */
	int x; /* key */
	int y; /* value */
	int r; /* index */
}; // 32 Byte

struct chunk_S{
	long t_ns; /* timestamp */
	int a; /* key */
	int b; /* value */
	int s; /* index */
}; // 32 Byte

struct ht{
	int counter;
	uint64_t address;
}; // 16 Byte

static const long n_sec = 1000000000L;

/* 
 * Calculate current timestamp
 */
__device__ inline
long r_get_tns(int generate_tuples_R, int r_iterations, int rate_R, int r, long *r_t_ns){
	if (r_iterations > 1) {
		return r_t_ns[r & (generate_tuples_R-1)]
			+ (r_iterations-1) * generate_tuples_R * (long)(1.e9 / rate_R);
	} else {
		return r_t_ns[r];
	}
}

__device__ inline
long s_get_tns(int generate_tuples_S, int s_iterations, int rate_S, int s, long *s_t_ns){
	if (s_iterations > 1) {
		return s_t_ns[s & (generate_tuples_S-1)]
			+ (s_iterations-1) * generate_tuples_S * (long)(1.e9 / rate_S);
	} else {
		return s_t_ns[s];
	}
}

/*
 * Includes: 
 *  1) Insertion of Tuple Block into HT S, 
 *  2) Comparision of Tuple Block with HT R, 
 *
 * Cleanup is launched in seperate kernel
 */
__global__ 
void compare_kernel_new_s_hj(
		int s_processed,
		int *output_buffer, int outsize_mask, 
		int *a, int *b, int *x, int *y, 
		long *s_ts, long *r_ts,
		int generate_tuples_S, int generate_tuples_R,
		int r_iterations, int s_iterations,
		int window_size_R, int window_size_S,
		int r_rate, int s_rate,
		ht *hmR, ht *hmS,
		unsigned *cleanup_bitmap_S, unsigned *cleanup_bitmap_R,
		int ht_size_r, int ht_size_s,
		int count, int* output_location,
		int *invalid_count_out) {
	unsigned int tid = threadIdx.x;
	/* const size_t idx = threadIdx.x + blockIdx.x * blockDim.x; */
	/* const long global_threads = blockDim.x * gridDim.x; */

    const int lane_id = threadIdx.x % warpSize;
    constexpr int leader_id = 0;
    const int warp_id = threadIdx.x / warpSize;
    const int num_warps = blockDim.x / warpSize;
    constexpr uint32_t warp_mask = 0xFFFFFFFFu;

    const int global_warp_id = blockIdx.x * num_warps + warp_id;
    const int global_warps = gridDim.x * num_warps;

	int invalid_count = 0;
	//if (idx == 0)
	//	invalid_count_out[0] = 0;
	extern __shared__ int sdata[];
        for (int s = global_warp_id + s_processed; s < count + s_processed; s += global_warps){

		const int k = a[s];
		//printf("add s: %d %d %d\n",k, s_processed, ht_size_s);

        // FIXME: parallelize with warp
        int hash = 0;
        if (lane_id == leader_id) {
            /* 
             * Build
             */

            /* get hash */
            MurmurHash_x86_32((void*)&k, sizeof(int), 0, &hash);

            hash = hash & (ht_size_s-1);
            int tpl_cntr = atomicAdd(&(hmS[hash].counter), 1);
            
            if (tpl_cntr >= 64) {
                printf("Chunk full at index: %d in S, hash: %d, s: %d \n", tpl_cntr, hash, s);
                __threadfence();
                assert(0);
            }

            chunk_S *chunk = (chunk_S*) hmS[hash].address;
            chunk[tpl_cntr].a = k;
            chunk[tpl_cntr].t_ns = s_get_tns(generate_tuples_S, s_iterations, s_rate, s, s_ts);
            chunk[tpl_cntr].b = b[s];
            chunk[tpl_cntr].s = s;
        }

		/* 
		 * Probe
		 */
		hash = __shfl_sync(warp_mask, hash, leader_id);
		int tpl_cntr = hmR[hash].counter;

		if (tpl_cntr != 0){
			const chunk_R *chunk = (chunk_R*) hmR[hash].address; // head
			for (int j = lane_id; j < tpl_cntr; j += warpSize){
				/*if (hash == 203630) {
					printf("%ld %ld\n", (chunk[j].t_ns + window_size_R * n_sec),
							s_get_tns(generate_tuples_S, s_iterations, s_rate, s, s_ts));
				}*/
				if ((chunk[j].t_ns + window_size_R * n_sec)
						> s_get_tns(generate_tuples_S, s_iterations, s_rate, s, s_ts)) { // Valid
					if (chunk[j].x == k) { // match
						int i = atomicAdd(output_location, 1) & outsize_mask;
						//printf("from s: %d %d\n",chunk[j].r, s);

						// Write into output buffer
						output_buffer[i*2]   = chunk[j].r;
						output_buffer[i*2+1] = s;
						
						//atomicAdd(num_out_tuples, 1);
					}
				} else { // Invalid
					set_bit(hash, cleanup_bitmap_R);
					atomicAdd(invalid_count_out, 1);
					//invalid_count_out[0]++;
					//invalid_count++;
				}
			}
		} 
	}

	/*sdata[tid] = invalid_count;
	__syncthreads();
	// do reduction in shared mem
	for(unsigned int s=1; s < blockDim.x; s *= 2) { 
		if (tid % (2*s) == 0) {
			sdata[tid] += sdata[tid + s]; 
		}
		__syncthreads(); 
	}
	// write result for this block to global mem
	if (tid == 0) 
		invalid_count_out[blockIdx.x] = sdata[0];*/
}

/*
 * Cleanup Kernel
 * launched in host code if cleanup threshold was reached
 * used threadnumber == ht_size
 */
__global__ 
void cleanup_r( int s_processed,
		int *output_buffer, int outsize_mask, 
		int *a, int *b, int *x, int *y, 
		long *s_ts, long *r_ts,
		int generate_tuples_S, int generate_tuples_R,
		int r_iterations, int s_iterations,
		int window_size_R, int window_size_S,
		int r_rate, int s_rate,
		ht *hmR, ht *hmS,
		unsigned *cleanup_bitmap_S, unsigned *cleanup_bitmap_R,
		int count) {
	const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	const long global_threads = blockDim.x * gridDim.x;

	for (int i = idx; i < count; i += global_threads){
		if (get_bit(i, cleanup_bitmap_R)){
			uint32_t tpl_cnt = hmR[i].counter;
			chunk_R *chunk = (chunk_R*) hmR[i].address; // head
			for(size_t j = 0; j < tpl_cnt; j++) { // non-empty
				// find first correct entry
				if ((chunk[j].t_ns + window_size_S * n_sec)
						> s_get_tns(generate_tuples_S, s_iterations, s_rate, s_processed, s_ts)) {
					// Remove + Move
					int l = 0;
					for(size_t u = j; u < tpl_cnt; u++, l++) { // non-empty
						chunk[l].t_ns = chunk[u].t_ns;
                                                chunk[l].x = chunk[u].x;
                                                chunk[l].y = chunk[u].y;
                                                chunk[l].r = chunk[u].r;
					}
					hmR[i].counter -= j;
					break;
				}

				// all entires are invalid
				if (tpl_cnt == j-1)
					hmR[i].counter = 0;
			}
			
		}
	}
}



__global__ 
void compare_kernel_new_r_hj(
		int r_processed,
		int *output_buffer, int outsize_mask, 
		int *a, int *b, int *x, int *y, 
		long *s_ts, long *r_ts,
		int generate_tuples_S, int generate_tuples_R,
		int r_iterations, int s_iterations,
		int window_size_R, int window_size_S,
		int r_rate, int s_rate,
		ht *hmR, ht *hmS,
		unsigned *cleanup_bitmap_S, unsigned *cleanup_bitmap_R,
		int ht_size_r, int ht_size_s,
		int count, int* output_location,
		int *invalid_count_out) {
	unsigned int tid = threadIdx.x;
	/* const size_t idx = threadIdx.x + blockIdx.x * blockDim.x; */
	/* const long global_threads = blockDim.x * gridDim.x; */

    const int lane_id = threadIdx.x % warpSize;
    constexpr int leader_id = 0;
    const int warp_id = threadIdx.x / warpSize;
    const int num_warps = blockDim.x / warpSize;
    constexpr uint32_t warp_mask = 0xFFFFFFFFu;

    const int global_warp_id = blockIdx.x * num_warps + warp_id;
    const int global_warps = gridDim.x * num_warps;

	//if (idx == 0)
	//	invalid_count_out[0] = 0;

	int invalid_count = 0;
	extern __shared__ int sdata[];
	for (int r = global_warp_id + r_processed; r < count + r_processed; r += global_warps){

		const int k = x[r];

        // FIXME: parallelize with warp
        int hash = 0;
        if (lane_id == leader_id) {
            /* 
             * Build
             */ 

            /* get hash */
            MurmurHash_x86_32((void*)&k, sizeof(int), 0, &hash);

            hash = hash & (ht_size_r-1);
            int tpl_cntr = atomicAdd(&(hmR[hash].counter), 1);
            
            if (tpl_cntr >= 64) {
                printf("%d\n", window_size_R);
                printf("Chunk full at index: %d in R, hash: %d, r: %d\n", tpl_cntr, hash, r);
                __threadfence();
                assert(0);
            }

            chunk_R *chunk = (chunk_R*) hmR[hash].address;
            chunk[tpl_cntr].x = k;
            chunk[tpl_cntr].t_ns = r_get_tns(generate_tuples_R, r_iterations, r_rate, r, r_ts);
            chunk[tpl_cntr].y = y[r];
            chunk[tpl_cntr].r = r;
        }

		/* 
		 * Probe
		 */
		hash = __shfl_sync(warp_mask, hash, leader_id);
        int tpl_cntr = hmS[hash].counter;

		if (tpl_cntr != 0){
			const chunk_S *chunk = (chunk_S*) hmS[hash].address; // head
			for (int j = lane_id; j < tpl_cntr; j += warpSize){
				if ((chunk[j].t_ns + window_size_S * n_sec)
						> r_get_tns(generate_tuples_R, r_iterations, r_rate, r, r_ts)) { // Valid
					if (chunk[j].a == k) { // match
						int i = atomicAdd(output_location, 1) & outsize_mask;

						// Write into output buffer 
						output_buffer[i*2]   = r;
						output_buffer[i*2+1] = chunk[j].s;
						
						//printf("from r: %d %d\n",r, chunk[j].s);
						//atomicAdd(num_out_tuples, 1);
					}
				} else { // Invalid
					set_bit(hash, cleanup_bitmap_S);
					atomicAdd(invalid_count_out, 1);
					//invalid_count_out[0]++;
					//invalid_count++;
				}
			}
		}
	}

	/*sdata[tid] = invalid_count;
	__syncthreads();
	// do reduction in shared mem
	for(unsigned int s=1; s < blockDim.x; s *= 2) { 
		if (tid % (2*s) == 0) {
			sdata[tid] += sdata[tid + s]; 
		}
		__syncthreads(); 
	}
	// write result for this block to global mem
	if (tid == 0) 
		invalid_count_out[blockIdx.x] = sdata[0];*/
		
}

/*
 * Cleanup Kernel
 * launched in host code if cleanup threshold was reached
 * used threadnumber == ht_size
 */
__global__ 
void cleanup_s(	int r_processed,
		int *output_buffer, int outsize_mask, 
		int *a, int *b, int *x, int *y, 
		long *s_ts, long *r_ts,
		int generate_tuples_S, int generate_tuples_R,
		int r_iterations, int s_iterations,
		int window_size_R, int window_size_S,
		int r_rate, int s_rate,
		ht *hmR, ht *hmS,
		unsigned *cleanup_bitmap_S, unsigned *cleanup_bitmap_R,
		int count) {
	const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	const long global_threads = blockDim.x * gridDim.x;

	for (int i = idx; i < count; i += global_threads){
		if (get_bit(i, cleanup_bitmap_S)){
			uint32_t tpl_cnt = hmS[i].counter;
			chunk_S *chunk = (chunk_S*) hmS[i].address; // head
			for(size_t j = 0; j < tpl_cnt; j++) { // non-empty
				// find first correct entry
				if ((chunk[j].t_ns + window_size_S * n_sec)
						> r_get_tns(generate_tuples_R, r_iterations, r_rate, r_processed, r_ts)) {
					// Remove + Move
					int l = 0;
					for(size_t u = j; u < tpl_cnt; u++,  l++) { // non-empty
						chunk[l].t_ns = chunk[u].t_ns;
                                                chunk[l].a = chunk[u].a;
                                                chunk[l].b = chunk[u].b;
                                                chunk[l].s = chunk[u].s;
					}
					hmS[i].counter -= j;
					//printf("%d\n",hmS[i].counter);
					break;
				}

				// all entires are invalid
				if (tpl_cnt == j-1)
					hmS[i].counter = 0;
			}
		}
	}
}

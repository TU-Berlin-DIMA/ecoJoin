#include "hip/hip_runtime.h"
#include <bitset>
#include <fstream>
#include <iostream>
#include <sstream>
#include <mutex>
#include <vector>
#include <queue>
#include <chrono>
#include <thread>
#include <cstring>
#include <omp.h>
#include "thread_pool.h"

#define MANAGED 0
#define ZERO_COPY_R 0

#define DEBUG_P(pr) if (DEBUG) std::cout << pr << "\n";

#define CUDA_SAFE(call)                                                 \
    do {                                                                \
	hipError_t err = call;                                             \
	if (hipSuccess != err) {                                           \
	    fprintf(stderr, "Cuda error in file '%s' in line %i : %s.",     \
		    __FILE__, __LINE__, hipGetErrorString(err));               \
	    exit(EXIT_FAILURE);                                             \
	}                                                                   \
    } while (0)

#define START_M(name) \
	std::chrono::time_point<std::chrono::system_clock> start_name, end_name; \
	start_name = std::chrono::system_clock::now(); 

#define END_M(name) \
	end_name = std::chrono::system_clock::now(); \
	int elapsed_seconds_name = std::chrono::duration_cast<std::chrono::milliseconds> (end_name-start_name).count(); \
    std::time_t end_time_name = std::chrono::system_clock::to_time_t(end_name);\
    std::cout << "elapsed time: " << elapsed_seconds_name << "ms\n";\

enum Stream { stream1, stream2 };

struct tuple {
    int key;
    int timestamp;
    int value;
};

struct record2 {
    int key;
    int timestamp;
    int left_value;
    int right_value;
};

tuple *compareTuples_s1_inter, *compareTuples_s1_comp;
__device__ size_t currentFIFO_s1 = 0;
size_t currentFIFO_s1_inter = 0;
tuple *compareTuples_s2_inter, *compareTuples_s2_comp;
__device__ size_t currentFIFO_s2 = 0;
size_t currentFIFO_s2_inter = 0;

std::vector<tuple> compareTuples_h;
int *compare_output_s1,  *compare_output_s2;
int *compare_output_prev;

int etpw, gridsize, blocksize;

std::ofstream myfile;

struct queueElm{
public:
	queueElm(int *compare_output, Stream stream, tuple new_tuple)
		: compare_output(compare_output), stream(stream), new_tuple(new_tuple)
	{}
private:
	int *compare_output;
	Stream stream;
	tuple new_tuple;
};

std::queue<queueElm> queue;

void printRecord(tuple rec){
	DEBUG_P("key: " << rec.key << " timestamp:  "<< rec.timestamp << " value: " << rec.value)
}

/* int per thread
 * no race conditions
 * idx = etpw / 32
 */
__global__ 
void compare_kernel_ipt(tuple input, int *output, size_t etpw, tuple *compareTuples) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	int z = 0;
	if ((idx+1) * 32 < etpw){
#pragma unroll
		for (int i = 0; i < 32; i++) {
			if  ((input.timestamp == compareTuples[idx+(32*i)].timestamp)  // Time Window
			        && (input.key == compareTuples[idx+(32*i)].key)) {
//				printf("%d %d \n", input.key, input.timestamp);
				z = z | 1 << i;
			}
		}
    } else if (idx * 32 < etpw){
		for (int i = 0; i < etpw - idx*32 ; i++) {
			if  ((input.timestamp == compareTuples[idx+(32*i)].timestamp)  // Time Window
			        && (input.key == compareTuples[idx+(32*i)].key)) {
//				printf("%d %d \n", input.key, input.timestamp);
				z = z | 1 << i;
			}
		}
	}

	output[idx] = z;
}

/*
 *  Adds a new tuple to the device storage
 *  Start as one thread
 */
__global__ 
void add_new_tuple_device(tuple new_tuple, Stream stream, size_t etpw, tuple *compareTuples) {
	if (stream == stream1) {
		compareTuples[currentFIFO_s1].key = new_tuple.key;
		compareTuples[currentFIFO_s1].value = new_tuple.value;
		compareTuples[currentFIFO_s1].timestamp = new_tuple.timestamp;
		//printf("add_new_tuple_device %d %d at %d \n", new_tuple.timestamp, new_tuple.key, currentFIFO_s1);
		if(++currentFIFO_s1 == etpw)
			currentFIFO_s1 = 0;
	} else {
		compareTuples[currentFIFO_s2].key = new_tuple.key;
		compareTuples[currentFIFO_s2].value = new_tuple.value;
		compareTuples[currentFIFO_s2].timestamp = new_tuple.timestamp;
		//printf("add_new_tuple_device %d %d at %d \n", new_tuple.timestamp, new_tuple.key, currentFIFO_s2);
		if(++currentFIFO_s2 == etpw)
			currentFIFO_s2 = 0;
	}
}

__global__ 
void print_state(tuple *compareTuples, int etpw) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) {
		for (int i = 0; i < etpw; i++)
			printf("(%d, %d) ", compareTuples[i].key, compareTuples[i].timestamp);
		printf("\n");
    }
}

/*
 *  Interprete Bitmap as joinResult
 */
std::vector<record2> interprete(tuple input, int *bitmap, Stream stream) {
    DEBUG_P("Current rec ")
	printRecord(input);

	// Add locally
	if (stream == stream1) {
		compareTuples_s1_inter[currentFIFO_s1_inter].key = input.key;
		compareTuples_s1_inter[currentFIFO_s1_inter].value = input.value;
		compareTuples_s1_inter[currentFIFO_s1_inter].timestamp = input.timestamp;
		currentFIFO_s1_inter++;
		if(currentFIFO_s1_inter == etpw)
			currentFIFO_s1_inter = 0;
	} else {
		compareTuples_s2_inter[currentFIFO_s2_inter].key = input.key;
		compareTuples_s2_inter[currentFIFO_s2_inter].value = input.value;
		compareTuples_s2_inter[currentFIFO_s2_inter].timestamp = input.timestamp;
		currentFIFO_s2_inter++;
		if(currentFIFO_s2_inter == etpw)
			currentFIFO_s2_inter = 0;
	}

    std::vector<record2> result;
    for (int i = 0; i < etpw; i = i + 32) {
		if (bitmap[i / 32] == 0) { // first check
			continue;
		} else {
#pragma unroll
			for (int k = 0; k < 32; k++){
				int j = i+k;
				if (std::bitset<32>(bitmap[j / 32]).test(j % 32)) { // fine-grained check
					//bitmap[i / 32] = bitmap[j / 32] & ~(1 << (j % 32));  // ith bit = 0
					record2 r;
					if (stream == stream1) {
						DEBUG_P( "Match  ")
						printRecord(input);
						printRecord(compareTuples_s2_inter[j]);
						
						r.key = compareTuples_s2_inter[j].key;
						r.left_value = compareTuples_s2_inter[j].value;
						r.right_value = input.value;
						r.timestamp = compareTuples_s2_inter[j].timestamp;
					} else { 
						DEBUG_P("Match  ")
						printRecord(input);
						printRecord(compareTuples_s1_inter[j]);

						r.key = compareTuples_s1_inter[j].key;
						r.left_value = compareTuples_s1_inter[j].value;
						r.right_value = input.value;
						r.timestamp = compareTuples_s1_inter[j].timestamp;
					}
					result.push_back(r);
				}
			}
			bitmap[i / 32] = 0;
		}
    }
	free(bitmap);
	return result;
}

void print_result(std::vector<record2> result) {
    for (auto a : result)
		std::cout << "match  newtuple (" << a.timestamp << ", " << a.key << ", "
		<< a.right_value << ", " << a.left_value << ") \n";
}

void write_result(std::vector<record2> result){
    for (auto a : result)
		myfile << "match  newtuple (" << a.timestamp << ", " << a.key << ", "
		<< a.right_value << ", " << a.left_value << ") \n";
}

void parseCSV(std::string filename, tuple *tup){
	std::ifstream file(filename);
	std::string line;
	int row = 0;
	while (std::getline(file, line)){
		std::stringstream iss(line);
		std::string key, time, val;
		std::getline(iss, key , ',');
		std::getline(iss, time, ',');
		std::getline(iss, val , ',');

		tup[row] = {std::stoi(key), std::stoi(time), std::stoi(val)};
		row++;
	}
	std::cout << filename << ": " << row << "rows loaded" << std::endl;
};

void workertask(int *compare_output, Stream stream, tuple new_tuple){
}

void startFileTest(std::string filename1, std::string filename2, int rows){;
	tuple *records0 = new tuple[rows];
	tuple *records1 = new tuple[rows];

	parseCSV(filename1.c_str(), records0);
	parseCSV(filename2.c_str(), records1);
	
    START_M(_)	
	int i = 0;
	int j = 0;
	Stream stream_prev;
	tuple  tuple_prev;
	tuple  new_tuple;

	hipStream_t cudastream1, cudastream2, cudastream3, cudastream4;
	hipStreamCreate(&cudastream1);
	hipStreamCreate(&cudastream2);
	hipStreamCreate(&cudastream3);
	hipStreamCreate(&cudastream4);
	
	ThreadPool pool(1);
	
    for (int k = 0; i < rows  && j < rows; k++) {
		while (records0[i].timestamp == k && i < rows) {
			new_tuple = records0[i];

			compare_kernel_ipt<<<blocksize, gridsize, 0, cudastream1>>>(new_tuple, compare_output_s1,  etpw, compareTuples_s2_comp);
			add_new_tuple_device<<<1, 1, 0, cudastream1>>>(new_tuple, stream1, etpw, compareTuples_s1_comp);

			//hipStreamSynchronize(cudastream1);
			hipStreamSynchronize(cudastream3);

			int *compare_output_new = (int *)malloc(((etpw / 32) + 1)* sizeof(int));
			std::memcpy(compare_output_new, compare_output_s1, sizeof(int) * ((etpw / 32) + 1));
			//queue.push(queueElm(compare_output_new, stream1, new_tuple));
			pool.enqueue([new_tuple, compare_output_new] (){write_result(interprete(new_tuple, compare_output_new, stream1));});

			i++;
			if (((i+j) % 10000) == 0) printf("%d\n", i+j);
		}
		while (records1[j].timestamp == k && j < rows) {
			new_tuple = records1[j];
			
			compare_kernel_ipt<<<blocksize, gridsize, 0, cudastream3>>>(new_tuple, compare_output_s2,  etpw, compareTuples_s1_comp);
			add_new_tuple_device<<<1, 1, 0, cudastream3>>>(new_tuple, stream2, etpw, compareTuples_s2_comp);

			hipStreamSynchronize(cudastream1);
			//hipStreamSynchronize(cudastream3);
			
			// No other kenel is invoced until the result is saved 
			int *compare_output_new = (int *)malloc(((etpw / 32) + 1)* sizeof(int));
			std::memcpy(compare_output_new, compare_output_s2, sizeof(int) * ((etpw / 32) + 1));
			//queue.push(queueElm(compare_output_new, stream2, new_tuple));
			pool.enqueue([new_tuple, compare_output_new] (){ write_result(interprete(new_tuple, compare_output_new, stream2));});

			j++;
			if (((i+j) % 10000) == 0) printf("%d\n", i+j);
		}
    }
	//write_result(interprete(tuple_prev, compare_output_prev, stream_prev));
	END_M(_)	
}



int main(int argc, char *argv[]){
	if (argc != 5){
		printf("Usage: hellsjoin_file [filename1] [filename2] [rows] [window]");
	}
	etpw = atoi(argv[4]);
	blocksize = 32;	           // Number of threads per block
	gridsize = (etpw / 1024) + 1;  // Number of blocks
	std::cout << "Blocksize: " << blocksize << " Gridsize: " << gridsize << "\n";
	//int minGridSize;
	//hipOccupancyMaxPotentialBlockSize(&minGridSize, &blocksize, compare_kernel_ipt, 0, (int)etpw / 32);
	//gridsize = (etpw  + blocksize - 1) / blocksize; 

#if MANAGED
    CUDA_SAFE(hipMallocManaged((void **)&compareTuples_s1_comp, sizeof(tuple) * etpw));
    CUDA_SAFE(hipMallocManaged((void **)&compareTuples_s2_comp, sizeof(tuple) * etpw));
    compareTuples_s1_inter = (tuple *)calloc(etpw, sizeof(tuple));
    compareTuples_s2_inter = (tuple *)calloc(etpw, sizeof(tuple));

    CUDA_SAFE(hipMallocManaged((void **)&compare_output_s1, sizeof(int) * ((etpw / 32) + 1)));
    CUDA_SAFE(hipMallocManaged((void **)&compare_output_s2, sizeof(int) * ((etpw / 32) + 1)));
    CUDA_SAFE(hipMallocManaged((void **)&compare_output_prev, sizeof(int) * ((etpw / 32) + 1)));
#elif ZERO_COPY_R
    compareTuples_s1_d = (tuple *)calloc(etpw, sizeof(tuple));
    CUDA_SAFE(hipHostRegister(compareTuples_s1_d, sizeof(tuple) * etpw, 2));
    compareTuples_s2_d = (tuple *)calloc(etpw, sizeof(tuple));
    CUDA_SAFE(hipHostRegister(compareTuples_s2_d, sizeof(tuple) * etpw, 2));

    compare_output = (int *)calloc(((etpw / 32) + 1), sizeof(int));
    CUDA_SAFE(hipHostRegister(compare_output, sizeof(int) * ((etpw / 32) + 1), 0));
#else  // ZERO_COPY_M
    CUDA_SAFE(hipMalloc((void **)&compareTuples_s1_comp, sizeof(tuple) * etpw));
    CUDA_SAFE(hipMalloc((void **)&compareTuples_s2_comp, sizeof(tuple) * etpw));
	
    //CUDA_SAFE(hipHostAlloc((void **)&compareTuples_s1_comp, sizeof(tuple) * etpw,0));
    //CUDA_SAFE(hipHostAlloc((void **)&compareTuples_s2_comp, sizeof(tuple) * etpw,0));
    compareTuples_s1_inter = (tuple *)calloc(etpw, sizeof(tuple));
    compareTuples_s2_inter = (tuple *)calloc(etpw, sizeof(tuple));
    
	//CUDA_SAFE(hipMalloc((void **)&compare_output_s1, sizeof(int) * ((etpw / 32) + 1)));
    //CUDA_SAFE(hipMalloc((void **)&compare_output_s2, sizeof(int) * ((etpw / 32) + 1)));

    CUDA_SAFE(hipHostAlloc((void **)&compare_output_s1, sizeof(int) * ((etpw / 32) + 1), 0));
    CUDA_SAFE(hipHostAlloc((void **)&compare_output_s2, sizeof(int) * ((etpw / 32) + 1), 0));
    CUDA_SAFE(hipHostAlloc((void **)&compare_output_prev, sizeof(int) * ((etpw / 32) + 1), 0));
#endif

    myfile.open ("result.csv");
#if FILE_
	startFileTest(argv[1],argv[2],atoi(argv[3])); // Filename1, Filename2, rows
#else
	startManualTest();    
#endif
    myfile.close();
}
